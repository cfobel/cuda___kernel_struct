#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "kernel_struct.hpp"
using namespace std;


class CUDApoint {
public:
    float a, b;
    __device__ void some_other_method() {}
};


template <class T>
__global__ void testKernel(T *data) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    data[i] = i;
}

template <>
__global__ void testKernel<point>(point *p) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    CUDApoint *test = (CUDApoint *)p;
    test[i].a = 1.1 * i;
    test[i].b = 2.2 * i;
}

template <class T>
vector<T> run_kernel(int num_points) {
    // set number of points 
    int gpuBlockSize = 4,
        gpuGridSize  = num_points / gpuBlockSize;

    // allocate memory
    vector<T> cpuPointArray(num_points);
    T *gpuPointArray;
    int mem_size = cpuPointArray.size() * sizeof(T);
    hipMalloc((void**)&gpuPointArray, mem_size);

    // launch kernel
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

    // retrieve the results
    hipMemcpy(&cpuPointArray[0], gpuPointArray, cpuPointArray.size() * sizeof(T), hipMemcpyDeviceToHost);
    // deallocate memory
    hipFree(gpuPointArray);
    return cpuPointArray;
}

vector<int> test_int(int num_points) {
    return run_kernel<int>(num_points);
}

vector<point> test_point(int num_points) {
    return run_kernel<point>(num_points);
}
